#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
using namespace std;
//CUDA RunTime API
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16




//�����������
void matgen(int* a, int n)
{
    int i, j;

    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {

            a[i * n + j] = (int)rand() / RAND_MAX + (int)rand() / (RAND_MAX * RAND_MAX);

        }
    }
}

void matgen(int* a, int c, int r)
{
    int i, j;

    for (i = 0; i < c; i++)
    {
        for (j = 0; j < r; j++)
        {

            a[i * c + j] = (int)rand() % 10;

        }
    }
}

template <size_t BLOCK_SIZE>
void __global__ MatMatMul(
    const int* A,
    const int* B,
    int* C,
    const size_t m,//a_r
    const size_t n,//b_r a_c
    const size_t k)//b_c
{
    int bx = blockIdx.x;		int by = blockIdx.y;
    int tx = threadIdx.x;		int ty = threadIdx.y;

    //ȷ����������е��к���
    int row = by * BLOCK_SIZE + ty;
    int column = bx * BLOCK_SIZE + tx;


    if (row < m && column < k)
    {
        int t = 0;

        for (int i = 0; i < n; i++)
        {
            t += A[row * n + i] * B[i * n + column];
        }
        C[row * n + column] = t;
    }

}







int main()
{



    //�������
    int* a, * b, * c, * d;

    const size_t n = 1 << 12;


    const size_t a_r = n, a_c = n, b_r = n, b_c = n;

    //�����ڴ�
    a = (int*)malloc(sizeof(int) * n * n);
    b = (int*)malloc(sizeof(int) * n * n);
    c = (int*)malloc(sizeof(int) * n * n);
    d = (int*)malloc(sizeof(int) * n * n);

    //�������������
    srand(0);

    //������ɾ���
    matgen(a, a_r, a_c);
    matgen(b, b_r, b_c);

    /*�����ݸ��Ƶ��Կ��ڴ���*/
    int* cuda_a, * cuda_b, * cuda_c;

    

    //hipMalloc ȡ��һ���Կ��ڴ� 
    hipMalloc((void**)&cuda_a, sizeof(int) * a_r * a_c);
    hipMalloc((void**)&cuda_b, sizeof(int) * b_r * b_c);
    hipMalloc((void**)&cuda_c, sizeof(int) * a_r * b_c);


    //hipMemcpy �������ľ����Ƶ��Կ��ڴ���
    //hipMemcpyHostToDevice - ���ڴ渴�Ƶ��Կ��ڴ�
    //hipMemcpyDeviceToHost - ���Կ��ڴ渴�Ƶ��ڴ�
    hipMemcpy(cuda_a, a, sizeof(int) * a_r * a_c, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(int) * b_r * b_c, hipMemcpyHostToDevice);



    //��������ṹ
    cout << "n1*n2:" << n << "*" << n << endl;
    cout << "TILE_WIDTH:" << TILE_WIDTH << endl;
    cout << "dimGrid:" << (b_c - 1) / TILE_WIDTH + 1 << ',' << (a_r - 1) / TILE_WIDTH + 1 << ',' << 1 << endl;
    cout << "dimBlock:" << TILE_WIDTH << ',' << TILE_WIDTH << ',' << 1 << endl;
    dim3 dimGrid((b_c - 1) / TILE_WIDTH + 1, (a_r - 1) / TILE_WIDTH + 1, 1);	//����ȡ��
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    clock_t sp = clock();
    cout << "start of CUDA:" << sp << endl;

    // ��CUDA ��ִ�к��� �﷨����������<<<block ��Ŀ, thread ��Ŀ, shared memory ��С>>>(����...);
    MatMatMul<TILE_WIDTH> << <dimGrid, dimBlock >> > (cuda_a, cuda_b, cuda_c, a_r, a_c/*b_r*/, b_c);


    //hipMemcpy ��������Դ��и��ƻ��ڴ�
    hipMemcpy(c, cuda_c, sizeof(int) * n * n, hipMemcpyDeviceToHost);
    //hipMemcpy(&time_use, time, sizeof(clock_t) * blocks_num * 2, hipMemcpyDeviceToHost);


    clock_t ep = clock();
    cout << "end of CUDA and start of CPU:" << ep << endl;
    cout << "cost:" << (double)(ep - sp) / CLOCKS_PER_SEC;
    //Free
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);
    hipFree(time);


//��֤��ȷ���뾫ȷ��
    /*
    //CPU����˷����������d
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            double t = 0;

            for (int k = 0; k < n; k++)
            {

                t += a[i * n + k] * b[k * n + j];

            }

            d[i * n + j] = t;

        }
    }
*/
    





    return 0;

}

